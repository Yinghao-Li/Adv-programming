#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "hip/hip_runtime_api.h"
#include "heat.h"
#include "hip/device_functions.h"


void Diffusion_2D(const float &k, const int &timestep, const float &start_temp, const int &width,
	const int &height, vector<ft_2D> &vec);
void Diffusion_3D(const float &k, const int &timestep, const float &start_temp, const int &width,
	const int &height, const int &depth, vector<ft_3D> &vec);


__global__ void New2Pre2D(float *pre_arr, const float *new_arr, dim3 dim) {
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x + dim.x;
	pre_arr[idx] = new_arr[idx];
}

__global__ void DiffusionCUDA2D(const float *pre_arr, float *new_arr, dim3 dim, const float k) {

	// Start from dim.x in case subscribes becomes negative
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x + dim.x;
	int idx_up = idx - dim.x;
	int idx_down = idx + dim.x;
	int idx_left = idx - 1;
	int idx_right = idx + 1;

	bool left_edge = idx % dim.x == 0;
	bool right_edge = (idx + 1) % dim.x == 0;
	bool top_edge = idx < 2 * dim.x;
	bool bottom_edge = (idx >= dim.x * dim.y) && (idx < dim.x * (dim.y + 1));

	if (idx < dim.x * (dim.y + 1)) {
		new_arr[idx] = pre_arr[idx] + k * (pre_arr[idx_up] * (!top_edge) + pre_arr[idx_down] * (!bottom_edge) +
			pre_arr[idx_left] * (!left_edge) + pre_arr[idx_right] * (!right_edge) - 4 * pre_arr[idx] +
			pre_arr[idx] * left_edge + pre_arr[idx] * right_edge + pre_arr[idx] * top_edge + pre_arr[idx] * bottom_edge);
	}
}

__global__ void RecoverFixed2D(float *new_arr, dim3 dim, int x, int y, int width, int height, float temp) {
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int offset_x = idx % width;
	int offset_y = idx / width;

	if (idx < width * height) {
		new_arr[(y + offset_y) * dim.x + x + offset_x] = temp;
	}
}


__global__ void New2Pre3D(float *pre_arr, const float *new_arr, dim3 dim) {
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x + dim.x * dim.y;
	pre_arr[idx] = new_arr[idx];
}


__global__ void DiffusionCUDA3D(const float *pre_arr, float *new_arr, dim3 dim, const float k) {

	// Start from dim.x in case subscribes becomes negative
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x + dim.x * dim.y;
	int idx_up = idx - dim.x;
	int idx_down = idx + dim.x;
	int idx_left = idx - 1;
	int idx_right = idx + 1;
	int idx_inner = idx - dim.x * dim.y;
	int idx_outer = idx + dim.x * dim.y;

	bool left_plane = idx % dim.x == 0;
	bool right_plane = (idx + 1) % dim.x == 0;
	bool upper_plane = idx % (dim.x * dim.y) < dim.x;
	bool bottom_plane = idx % (dim.x * dim.y) >= dim.x * (dim.y - 1);
	bool inner_plane = idx < 2 * dim.x * dim.y;
	bool outer_plane = (idx >= dim.x * dim.y * dim.z) && (idx < dim.x * dim.y * (dim.z + 1));

	if (idx < dim.x * dim.y * (dim.z + 1)) {
		new_arr[idx] = pre_arr[idx] + k * (pre_arr[idx_up] * (!upper_plane) + pre_arr[idx_down] * (!bottom_plane) +
			pre_arr[idx_left] * (!left_plane) + pre_arr[idx_right] * (!right_plane) +
			pre_arr[idx_inner] * (!inner_plane) + pre_arr[idx_outer] * (!outer_plane) - 
			6 * pre_arr[idx] + pre_arr[idx] * left_plane + pre_arr[idx] * right_plane + 
			pre_arr[idx] * upper_plane + pre_arr[idx] * bottom_plane + pre_arr[idx] * inner_plane + 
			pre_arr[idx] * outer_plane);
	}
}


__global__ void RecoverFixed3D(float *new_arr, dim3 dim, int x, int y, int z, int width, int height, int depth, float temp) {
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int offset_x = (idx % (height * width)) % width;
	int offset_y = (idx % (height * width)) / width;
	int offset_z = idx / (height * width);

	if (idx < width * height * depth) {
		new_arr[(z + offset_z) * dim.x * dim.y + (y + offset_y) * dim.x + x + offset_x] = temp;
	}
}


ft_2D * parseVec(const vector<ft_2D> &vec) {
	ft_2D * p = (ft_2D*)malloc(sizeof(ft_2D) * vec.size());
	for (int i = 0; i < vec.size(); ++i) {
		p[i] = vec[i];
	}
	return p;
}

ft_3D * parseVec(const vector<ft_3D> &vec) {
	ft_3D * p = (ft_3D*)malloc(sizeof(ft_3D) * vec.size());
	for (int i = 0; i < vec.size(); ++i) {
		p[i] = vec[i];
	}
	return p;
}


int main(int argc, char** argv) {
	// Argument Parsing
	if (argc == 1) {
		cout << "\nNo argument was passed.\n";
		exit(1);
	}
	else if (argc != 2) {
		cout << "\nThe number of argument is incorrect.\n";
		exit(1);
	}

	// Initial parameters
	float k = 0.0;
	int timestep = 0;
	int width = 0;
	int height = 0;
	int depth = 0;
	float start_temp = 0.0;
	vector<ft_2D> vec_2D;
	vector<ft_3D> vec_3D;
	bool mode;

	// Get Parameters
	mode = ReadFile(argv[1], k, timestep, start_temp, width, height, depth, vec_2D, vec_3D);

	// Calculating
	if (mode == D_2) {
		Diffusion_2D(k, timestep, start_temp, width, height, vec_2D);
	}
	else {
		Diffusion_3D(k, timestep, start_temp, width, height, depth, vec_3D);
	}

	cout << "Compute Finished" << endl;

    return 0;
}


// Function to manipulate 2D heat diffusion
void Diffusion_2D(const float &k, const int &timestep, const float &start_temp, const int &width,
	const int &height, vector<ft_2D> &vec) {

	ft_2D * tvec = parseVec(vec);

	// extend graph
	dim3 dim(width, height, 1);
	unsigned int length = dim.x * dim.y;
	unsigned int mem_size = sizeof(float) * (length + dim.x);

	// Allocate memory for graphs.
	float *pre_graph = reinterpret_cast<float *>(malloc(mem_size));
	float *new_graph = reinterpret_cast<float *>(malloc(mem_size));

	// Construct initial graph with start temperature and heat source.
	for (int i = 0; i < dim.y + 1; ++i) {
		for (int j = 0; j < dim.x; ++j) {
			pre_graph[i * dim.x + j] = start_temp;
		}
	}
	for (vector<ft_2D>::iterator iter = vec.begin(); iter != vec.end(); ++iter) {
		for (int i = iter->l_y + 1; i < iter->l_y + 1 + iter->height; ++i) {
			for (int j = iter->l_x; j < iter->l_x + iter->width; ++j) {
				pre_graph[i * dim.x + j] = iter->fixed_temp;
			}
		}
	}
	int vec_size = vec.size();

	// Allocate device memory
	float *d_pre, *d_new;

	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_pre), mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_new), mem_size));

	// copy host memory to device
	checkCudaErrors(hipMemcpy(d_pre, pre_graph, mem_size, hipMemcpyHostToDevice));

	// Create and start timer
	printf("Computing result using CUDA Kernel...\n");

	// Compute new temperatures for t iteration.
	for (int t = 0; t < timestep; ++t) {
		DiffusionCUDA2D <<< (length + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (d_pre, d_new, dim, k);
		for (int i = 0; i < vec_size; ++i) {
			RecoverFixed2D <<< (vec[i].width * vec[i].height + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (d_new, dim,
				vec[i].l_x, vec[i].l_y + 1, vec[i].width, vec[i].height, vec[i].fixed_temp);
		}
		New2Pre2D <<< (length + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (d_pre, d_new, dim);
	}

	// Copy result from device to host
	checkCudaErrors(hipMemcpy(new_graph, d_new, mem_size, hipMemcpyDeviceToHost));

	// Write file
	ofstream f_out;
	f_out.open("heat2Doutput.csv", ios::out);
	assert(f_out);
	for (int i = 1; i < height + 1; ++i) {
		for (int j = 0; j < width - 1; ++j) {
			f_out << new_graph[i * width + j] << ", ";
		}
		if (i != height) f_out << new_graph[i * width + width - 1] << endl;
		else f_out << new_graph[i * width + width - 1];
	}

	// Close file
	f_out.close();

	// Clean up memory
	free(new_graph);
	free(pre_graph);
	checkCudaErrors(hipFree(d_pre));
	checkCudaErrors(hipFree(d_new));
}



// Function to manipulate 3D heat diffusion
void Diffusion_3D(const float &k, const int &timestep, const float &start_temp, const int &width,
	const int &height, const int &depth, vector<ft_3D> &vec) {

	ft_3D * tvec = parseVec(vec);

	// extend graph
	dim3 dim(width, height, depth);
	unsigned int length = dim.x * dim.y * dim.z;
	unsigned int mem_size = sizeof(float) * (length + dim.x * dim.y);

	// Allocate memory for graphs.
	float *pre_graph = reinterpret_cast<float *>(malloc(mem_size));
	float *new_graph = reinterpret_cast<float *>(malloc(mem_size));

	// Construct initial graph with start temperature and heat source.
	for (int m = 0; m < depth + 1; ++m) {
		for (int i = 0; i < height; ++i) {
			for (int j = 0; j < width; ++j) {
				pre_graph[m * height * width + i * width + j] = start_temp;
			}
		}
	}
	for (vector<ft_3D>::iterator iter = vec.begin(); iter != vec.end(); ++iter) {
		for (int m = iter->l_z + 1; m < iter->l_z + iter->depth + 1; ++m) {
			for (int i = iter->l_y; i < iter->l_y + iter->height; ++i) {
				for (int j = iter->l_x; j < iter->l_x + iter->width; ++j) {
					pre_graph[m * height * width + i * width + j] = iter->fixed_temp;
				}
			}
		}
	}
	
	int vec_size = vec.size();

	// Allocate device memory
	float *d_pre, *d_new;

	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_pre), mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_new), mem_size));

	// copy host memory to device
	checkCudaErrors(hipMemcpy(d_pre, pre_graph, mem_size, hipMemcpyHostToDevice));

	// Create and start timer
	printf("Computing result using CUDA Kernel...\n");

	// Compute new temperatures for t iteration.
	for (int t = 0; t < timestep; ++t) {
		DiffusionCUDA3D <<< (length + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (d_pre, d_new, dim, k);
		for (int i = 0; i < vec_size; ++i) {
			RecoverFixed3D <<< (vec[i].width * vec[i].height * vec[i].depth + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
				d_new, dim, vec[i].l_x, vec[i].l_y, vec[i].l_z + 1, vec[i].width, vec[i].height,
				vec[i].depth, vec[i].fixed_temp);
		}
		New2Pre3D <<< (length + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (d_pre, d_new, dim);
	}

	// Copy result from device to host
	checkCudaErrors(hipMemcpy(new_graph, d_new, mem_size, hipMemcpyDeviceToHost));

	// Write file
	ofstream f_out;
	f_out.open("heat3Doutput.csv", ios::out);
	assert(f_out);
	for (int m = 1; m < depth + 1; ++m) {
		for (int i = 0; i < height; ++i) {
			for (int j = 0; j < width - 1; ++j) {
				f_out << new_graph[m * height * width + i * width + j] << ", ";
			}
			if (m == depth && i == height - 1) f_out << new_graph[m * height * width + i * width + width - 1];
			else f_out << new_graph[m * height * width + i * width + width - 1] << endl;
		}
		if (m != depth) f_out << endl;
	}

	//for (int m = 1; m < depth + 1; ++m) {
	//	for (int i = 0; i < height; ++i) {
	//		for (int j = 0; j < width - 1; ++j) {
	//			f_out << pre_graph[m * height * width + i * width + j] << ", ";
	//		}
	//		if (m == depth && i == height) f_out << pre_graph[m * height * width + i * width + width - 1];
	//		else f_out << pre_graph[m * height * width + i * width + width - 1] << endl;
	//	}
	//	if (m != depth) f_out << endl;
	//}

	// Close file
	f_out.close();

	// Clean up memory
	free(new_graph);
	free(pre_graph);
	checkCudaErrors(hipFree(d_pre));
	checkCudaErrors(hipFree(d_new));
}
